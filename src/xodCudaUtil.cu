/*------------------------------------------------------------------------------------------------*/
/* ___::((xodCudaUtil.cu))::___

   ___::((created by eschei))___

	Purpose: CMake CUDA Accelerated Image experiments

	Revision History: 2024-04-27 - initial
*/

/*------------------------------------------------------------------------------------------------*/

#include <iostream>
#include <hip/hip_runtime.h>

#include "../include/xodCudaUtil.h"



void checkCudaErrors(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}